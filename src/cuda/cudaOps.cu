#include "kernels/utils.cuh"
#include "kernels/slice_cols.cu"




void CudaOps::add(const double* h_A, const double* h_B, double* h_C, size_t n) {
    double *d_A, *d_B, *d_C;
    size_t bytes = n * sizeof(double);

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    LaunchConfig cfg = make_launch_1d(n);
    add_forward_kernel<<<cfg.blocks, cfg.threads>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void CudaOps::add_broadcast(const double* h_A, const double* h_B, double* h_C, int M, int N) {
    size_t size_A = M * N * sizeof(double);
    size_t size_B = N * sizeof(double);
    size_t size_C = M * N * sizeof(double);

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    LaunchConfig cfg = make_launch_1d(M * N);
    add_broadcast_forward_kernel<<<cfg.blocks, cfg.threads>>>(d_A, d_B, d_C, M, N);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


void CudaOps::add_backward(const double* h_dC, double* h_dA, double* h_dB, size_t n) {
    double *d_dA, *d_dB, *d_dC;
    size_t bytes = n * sizeof(double);

    hipMalloc(&d_dA, bytes);
    hipMalloc(&d_dB, bytes);
    hipMalloc(&d_dC, bytes);

    hipMemcpy(d_dC, h_dC, bytes, hipMemcpyHostToDevice);

    LaunchConfig cfg = make_launch_1d(n);
    add_backward_kernel<<<cfg.blocks, cfg.threads>>>(d_dA, d_dB, d_dC, n);
    hipDeviceSynchronize();

    hipMemcpy(h_dA, d_dA, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_dB, d_dB, bytes, hipMemcpyDeviceToHost);

    hipFree(d_dA);
    hipFree(d_dB);
    hipFree(d_dC);
}

void CudaOps::add_broadcast_backward(double* h_dA, double* h_dB, const double* h_dC, int M, int N) {
    size_t size_A = M * N * sizeof(double);
    size_t size_B = N * sizeof(double);
    size_t size_C = M * N * sizeof(double);

    double *d_dA, *d_dB, *d_dC;
    hipMalloc(&d_dA, size_A);
    hipMalloc(&d_dB, size_B);
    hipMalloc(&d_dC, size_C);

    hipMemcpy(d_dC, h_dC, size_C, hipMemcpyHostToDevice);
    hipMemset(d_dB, 0, size_B); 

    auto [blocks, threads] = make_launch_1d(M * N);
    add_broadcast_backward_kernel<<<blocks, threads>>>(d_dA, d_dB, d_dC, M, N);
    hipDeviceSynchronize();

    hipMemcpy(h_dA, d_dA, size_A, hipMemcpyDeviceToHost);
    hipMemcpy(h_dB, d_dB, size_B, hipMemcpyDeviceToHost);

    hipFree(d_dA);
    hipFree(d_dB);
    hipFree(d_dC);
}



void CudaOps::gelu(const double* h_A, double* h_C, size_t n) {
    //op= C=gelu(X) X= MxN | C=MxN
    double *d_A, *d_C;
    size_t bytes = n * sizeof(double);

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);


    LaunchConfig cfg = make_launch_1d(n);
    gelu_forward_kernel<<<cfg.blocks, cfg.threads>>>(d_A, d_C, n);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_C);
}



void CudaOps::gelu_backward(const double* h_A, const double* h_dOut, double* h_dX, size_t n) {
    size_t bytes = n * sizeof(double);
    double *d_A, *d_dOut, *d_dX;

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_dOut, bytes);
    hipMalloc(&d_dX, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_dOut, h_dOut, bytes, hipMemcpyHostToDevice);

    LaunchConfig cfg = make_launch_1d(n);
    gelu_backward_kernel<<<cfg.blocks, cfg.threads>>>(d_A, d_dOut, d_dX, n);
    hipDeviceSynchronize();

    hipMemcpy(h_dX, d_dX, bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_dOut);
    hipFree(d_dX);
}



void CudaOps::slice_cols(
    const double* h_A, double* h_C,
    int M, int N, int start_col, int width
) {
    size_t size_A = M * N * sizeof(double);
    size_t size_C = M * width * sizeof(double);

    double *d_A, *d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);

    LaunchConfig cfg = make_launch_2d(M, width);
    slice_cols_forward_kernel<<<cfg.blocks, cfg.threads>>>(d_A, d_C, M, N, start_col, width);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_C);
}

void CudaOps::slice_cols_backward(
    const double* h_grad_out, double* h_grad_in,
    int M, int N, int start_col, int width
) {
    size_t size_grad_out = M * width * sizeof(double);
    size_t size_grad_in  = M * N * sizeof(double);

    double *d_grad_out, *d_grad_in;
    hipMalloc(&d_grad_out, size_grad_out);
    hipMalloc(&d_grad_in,  size_grad_in);

    hipMemcpy(d_grad_out, h_grad_out, size_grad_out, hipMemcpyHostToDevice);
    hipMemset(d_grad_in, 0, size_grad_in);

    LaunchConfig cfg = make_launch_2d(M, width);
    slice_cols_backward_kernel<<<cfg.blocks, cfg.threads>>>(d_grad_out, d_grad_in, M, N, start_col, width);
    hipDeviceSynchronize();

    hipMemcpy(h_grad_in, d_grad_in, size_grad_in, hipMemcpyDeviceToHost);

    hipFree(d_grad_out);
    hipFree(d_grad_in);
}




void CudaOps::matmul(const double* h_A, const double* h_B, double* h_C,
                     int M, int N, int K) {
    size_t size_A = M * K * sizeof(double);
    size_t size_B = K * N * sizeof(double);
    size_t size_C = M * N * sizeof(double);

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    LaunchConfig cfg = make_launch_2d(M, N, 16);
    matmul_forward_kernel<<<cfg.blocks, cfg.threads>>>(d_A, d_B, d_C, M, N, K);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
void CudaOps::matmul_backward(const double* h_A, const double* h_B,
                              const double* h_grad_C,
                              double* h_grad_A, double* h_grad_B,
                              int M, int N, int K) {
    size_t size_A = M * K * sizeof(double);
    size_t size_B = K * N * sizeof(double);
    size_t size_C = M * N * sizeof(double);

    double *d_A, *d_B, *d_grad_C, *d_grad_A, *d_grad_B;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_grad_C, size_C);
    hipMalloc(&d_grad_A, size_A);
    hipMalloc(&d_grad_B, size_B);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    hipMemcpy(d_grad_C, h_grad_C, size_C, hipMemcpyHostToDevice);

    // grad_A = grad_C * B^T
    LaunchConfig cfgA = make_launch_2d(M, K, 16);
    matmul_backward_A_kernel<<<cfgA.blocks, cfgA.threads>>>(d_grad_C, d_B, d_grad_A, M, K, N);

    // grad_B = A^T * grad_C
    LaunchConfig cfgB = make_launch_2d(K, N, 16);
    matmul_backward_B_kernel<<<cfgB.blocks, cfgB.threads>>>(d_A, d_grad_C, d_grad_B, M, K, N);

    hipDeviceSynchronize();

    hipMemcpy(h_grad_A, d_grad_A, size_A, hipMemcpyDeviceToHost);
    hipMemcpy(h_grad_B, d_grad_B, size_B, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_grad_C);
    hipFree(d_grad_A);
    hipFree(d_grad_B);
}


void CudaOps::mul_broadcast(
    const double* h_A, const double* h_B, double* h_C,
    int M, int N
) {
    size_t bytes_A = M * N * sizeof(double);
    size_t bytes_B = N * sizeof(double);

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes_A);
    hipMalloc(&d_B, bytes_B);
    hipMalloc(&d_C, bytes_A);

    hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes_B, hipMemcpyHostToDevice);

    LaunchConfig cfg = make_launch_1d(M * N);
    mul_broadcast_forward_kernel<<<cfg.blocks, cfg.threads>>>(d_A, d_B, d_C, M, N);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, bytes_A, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void CudaOps::mul_broadcast_backward(
    const double* h_A, const double* h_B, const double* h_dOut,
    double* h_dA, double* h_dB,
    int M, int N
) {
    size_t bytes_A = M * N * sizeof(double);
    size_t bytes_B = N * sizeof(double);

    double *d_A, *d_B, *d_dOut, *d_dA, *d_dB;
    hipMalloc(&d_A, bytes_A);
    hipMalloc(&d_B, bytes_B);
    hipMalloc(&d_dOut, bytes_A);
    hipMalloc(&d_dA, bytes_A);
    hipMalloc(&d_dB, bytes_B);

    hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes_B, hipMemcpyHostToDevice);
    hipMemcpy(d_dOut, h_dOut, bytes_A, hipMemcpyHostToDevice);
    hipMemset(d_dB, 0, bytes_B); 

    LaunchConfig cfg = make_launch_1d(M * N);
    mul_broadcast_backward_kernel<<<cfg.blocks, cfg.threads>>>(
        d_A, d_B, d_dOut, d_dA, d_dB, M, N
    );
    hipDeviceSynchronize();

    hipMemcpy(h_dA, d_dA, bytes_A, hipMemcpyDeviceToHost);
    hipMemcpy(h_dB, d_dB, bytes_B, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_dOut);
    hipFree(d_dA);
    hipFree(d_dB);
}


void CudaOps::relu(const double* h_A, double* h_C, size_t n) {
    //op= C=relu(X) X= MxN | C=MxN
    double *d_A, *d_C;
    size_t bytes = n * sizeof(double);

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);


    LaunchConfig cfg = make_launch_1d(n);
    relu_forward_kernel<<<cfg.blocks, cfg.threads>>>(d_A, d_C, n);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_C);
}


void CudaOps::relu_backward(const double* h_A, const double* h_dOut, double* h_dX, size_t n) {
    size_t bytes = n * sizeof(double);
    double *d_A, *d_dOut, *d_dX;

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_dOut, bytes);
    hipMalloc(&d_dX, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_dOut, h_dOut, bytes, hipMemcpyHostToDevice);

    LaunchConfig cfg = make_launch_1d(n);
    relu_backward_kernel<<<cfg.blocks, cfg.threads>>>(d_A, d_dOut, d_dX, n);
    hipDeviceSynchronize();

    hipMemcpy(h_dX, d_dX, bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_dOut);
    hipFree(d_dX);
}



void CudaOps::scale_backward(
    const double* h_dC,  
    double* h_dA,       
    int M, int N,
    double scale
) {
    int size = M * N;
    size_t bytes = size * sizeof(double);

    double *d_dC, *d_dA;
    hipMalloc(&d_dC, bytes);
    hipMalloc(&d_dA, bytes);

    hipMemcpy(d_dC, h_dC, bytes, hipMemcpyHostToDevice);

    LaunchConfig cfg = make_launch_1d(size);
    scale_backward_kernel<<<cfg.blocks, cfg.threads>>>(d_dC, d_dA, scale, size);
    hipDeviceSynchronize();

    hipMemcpy(h_dA, d_dA, bytes, hipMemcpyDeviceToHost);

    hipFree(d_dC);
    hipFree(d_dA);
}

void CudaOps::scale(const double* h_A, double* h_C, int M, int N, double scale) {
    int numElements = M * N;
    size_t size = numElements * sizeof(double);

    double *d_A, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    LaunchConfig cfg = make_launch_1d(numElements);
    scale_forward_kernel<<<cfg.blocks, cfg.threads>>>(d_A, d_C, scale, numElements);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_C);
}


void CudaOps::transpose(const double* h_A, double* h_C, int M, int N) {
    size_t size_A = M * N * sizeof(double);
    size_t size_C = N * M * sizeof(double);

    double *d_A, *d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);

    LaunchConfig cfg = make_launch_2d(M, N);
    transpose_forward_kernel<<<cfg.blocks, cfg.threads>>>(d_A, d_C, M, N);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_C);
}


void CudaOps::transpose_backward(const double* h_dOut, double* h_dX, int M, int N) {
    size_t size = M * N * sizeof(double);

    double *d_dOut, *d_dX;
    hipMalloc(&d_dOut, size);
    hipMalloc(&d_dX, size);

    hipMemcpy(d_dOut, h_dOut, size, hipMemcpyHostToDevice);

    LaunchConfig cfg = make_launch_2d(M, N);
    transpose_backward_kernel<<<cfg.blocks, cfg.threads>>>(d_dOut, d_dX, M, N);
    hipDeviceSynchronize();

    hipMemcpy(h_dX, d_dX, size, hipMemcpyDeviceToHost);

    hipFree(d_dOut);
    hipFree(d_dX);
}
